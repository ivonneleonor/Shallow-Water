#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipsparse.h"
#include "hipblas.h"
#include "hello.cuh"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define CUDA_CALL(x) do {if((x)!=hipSuccess){\
   printf("Error at %s:%d\n",__FILE__,__LINE__);\
   }}while(0)

#define CLEANUP(s)      \
do{                     \
   printf("%s\n",s);    \
   if(I)     free(I);   \
   if(J)     free(J);   \
   if(val)   free(val); \
   if(r)     free(r);   \
   if(d_col)  hipFree(d_col);   \
   if(d_row)  hipFree(d_row);   \
   if(d_val)  hipFree(d_val);   \
   if(d_x)    hipFree(d_x);     \
   if(d_r)    hipFree(d_r);     \
   if(d_p)    hipFree(d_p);     \
   if(d_Ax)   hipFree(d_Ax);    \
   if(d_r_c)  hipFree(d_r_c);   \
   if(d_p_c)  hipFree(d_p_c);   \
   if(descr)          hipsparseDestroyMatDescr(descr);  \
   if(cublasHandle)   hipblasDestroy(cublasHandle);   \
   if(cusparseHandle) hipsparseDestroy(cusparseHandle);   \
   hipDeviceReset(); \
   fflush(stdout); \
} while(0)
/*
__global__ void setup_kernel(hiprandState *state)
{
  int id=threadIdx.x+blockIdx.x*64;
  hiprand_init(1234,id,0,&state[id]);
}

__global__ void generate_kernel(hiprandState *state, unsigned int *result,unsigned int N)
{
  int id=threadIdx.x+blockIdx.x*64;
  int count=0;
  unsigned int x;
  hiprandState localState=state[id];
  for(int n=0;n<N;n++)
    { x=hiprand(&localState);}
  if((x&1))
    {count++;}

  state[id]=localState;
  result[id]+=count;
}



__global__ void set(double *dx,int N)
{
 int tid=threadIdx.x+blockIdx.x*64;
 if (tid<N)
i dx[tid]=1.0;
}
*/
extern "C"

{
 double *solverbicg(int* J,double* val,int* I,double* x,double* r, int N,int nz)
{

FILE *p1;
FILE *p2;
p1=fopen("Ax.dat","w+");
p2=fopen("x.dat","w+");
int i, k, kmax=4500;
const float tol = 1e-3f;
double alpha=1.0, alpham1 = -1.0, beta=0.0, r0=0.0, r1=0.0, dot=0.0, a=0.0, na=0.0, b=0.0;
double *d_Ax=0, *test=0,*test1=0, *d_val=0,*d_x=0, *d_r=0, *d_p=0, *d_r_c=0, *d_p_c=0;
int *d_col=0, *d_row=0;
//unsigned int total=0;
 unsigned int  *hostResults;
hipError_t cudaStat1,cudaStat2,cudaStat3, cudaStat4,cudaStat5,cudaStat6,cudaStat7;
hipblasStatus_t cublasStatus1;
hipblasHandle_t cublasHandle=0;
hipsparseStatus_t cusparseStatus1;
hipsparseHandle_t cusparseHandle;
hipsparseMatDescr_t descr;

   for(i=0;i<10;i++){
       printf("i=%d,J=%d,val=%7.3f,I=%d,x=%7.3f ,r=%e \n",i,J[i],val[i],I[i],x[i],r[i]);}
   printf("N=%d  nz=%d\n",N,nz);


hostResults=(unsigned int *)calloc(N,sizeof(int));
test = (double *)malloc(sizeof(double)*N);
test1 = (double *)malloc(sizeof(double)*N);
if((!test)||(!test1)||(!hostResults)){CLEANUP("Memory on host failed,test\n");}
 
    cudaStat1=hipMalloc((void **)&d_col, nz*sizeof(int));
    cudaStat2=hipMalloc((void **)&d_row, (N+1)*sizeof(int));
    cudaStat3=hipMalloc((void **)&d_val, nz*sizeof(double));
    cudaStat4=hipMalloc((void **)&d_x, N*sizeof(double));
    cudaStat5=hipMalloc((void **)&d_r, N*sizeof(double));
    cudaStat6=hipMalloc((void **)&d_p, N*sizeof(double));
    cudaStat7=hipMalloc((void **)&d_Ax, N*sizeof(double));
   if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat3!=hipSuccess)||(cudaStat4!=hipSuccess)||(cudaStat5!=hipSuccess)||(cudaStat6!=hipSuccess)||(cudaStat7!=hipSuccess)){printf("Memcpy from Host to Device failed"); }

// CUDA_CALL(hipMalloc((void **)&devResults,N*sizeof(unsigned int)));

// CUDA_CALL(hipMemset(d_x,1,N*sizeof(unsigned int)));
/*
 CUDA_CALL(hipMalloc((void **)&devStates,N*sizeof(hiprandState)));
 generate_kernel<<<64,64>>>(devStates,devResults,N);
 CUDA_CALL(hipMemcpy(hostResults,devResults,N*sizeof(unsigned int),hipMemcpyDeviceToHost));

 for(i=0;i<N;i++)
 {
    total+=hostResults[i];
    printf("%d",hostResults[i]);
 }
 printf("Total de unos=%d\n",total);

for(i=0;i<nz;i++){
    printf("i=%d,J=%d,val=%7.3f,I=%d r=%7.3f \n",i,J[i],val[i],I[i],r[i]);}
 printf("N=%d  nz=%d\n",N,nz);
*/

 /* set<<<64,64>>>(d_x,N);

CUDA_CALL(hipMemcpy(test,d_x,N*sizeof(double),hipMemcpyDeviceToHost));
printf("d_x de la copia \n");

 for(i=0;i<N;i++)
 {
  //  total+=hostResults[i];
    printf("%f\n",test[i]);
 }

*/
/*
printf("desde el solver");
for(i=0;i<10;i++)
printf("J[%d]=%d\n",i,J[i]);
*/
    cudaStat1=hipMemcpy(d_col, J, nz*sizeof(int), hipMemcpyHostToDevice);
    cudaStat2=hipMemcpy(d_row, I, (N+1)*sizeof(int), hipMemcpyHostToDevice);
    cudaStat3=hipMemcpy(d_val, val, nz*sizeof(double), hipMemcpyHostToDevice);
    cudaStat4=hipMemcpy(d_x, x, N*sizeof(double), hipMemcpyHostToDevice);
    cudaStat5=hipMemcpy(d_r, r, N*sizeof(double), hipMemcpyHostToDevice);
    if((cudaStat1!=hipSuccess)||(cudaStat2!=hipSuccess)||(cudaStat3!=hipSuccess)||(cudaStat4!=hipSuccess)||(cudaStat5!=hipSuccess)){printf("Memcpy from Host to Device failed\n");}

cusparseStatus1=hipsparseCreate(&cusparseHandle);
 if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
   CLEANUP("Cusparse create handle failed\n");}

cublasStatus1=hipblasCreate(&cublasHandle);
  if(cublasStatus1!=HIPBLAS_STATUS_SUCCESS){
     CLEANUP("Cublas create handle failed \n");}


cusparseStatus1=hipsparseCreateMatDescr(&descr);
  if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
        printf("Descriptor creation failed\n");}//Set matrix type and index base

hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

//printf("Start solver =)");

 cusparseStatus1=hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("Ax0 performing failed\n");}

  hipblasDaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    if(cublasStatus1!=HIPBLAS_STATUS_SUCCESS){
     CLEANUP("b-Ax0 performing failed\n"); }


cublasStatus1 = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);


k=1;

 while (r1 > tol*tol && k <= kmax)
    {
        if (k > 1)
        {
            b = r1 / r0;
            cublasStatus1 = hipblasDscal(cublasHandle, N, &b, d_p, 1);
            cublasStatus1 = hipblasDaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
        }
        else
        {
            cublasStatus1 = hipblasDcopy(cublasHandle, N, d_r, 1, d_p, 1);
        }

        hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
        cublasStatus1 = hipblasDdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
        a = r1 / dot;

        cublasStatus1 = hipblasDaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
        na = -a;
        cublasStatus1 = hipblasDaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

        r0 = r1;
        cublasStatus1 = hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
        hipDeviceSynchronize();
//        printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
        k++;
    }

//while(r1!=0.0 && k<kmax);

/*  
cudaStat1=hipMemcpy(test,  d_x, N*sizeof(double), hipMemcpyDeviceToHost);
  if(cudaStat1!=hipSuccess){
    CLEANUP("Memcpy from Device to Host failed\n"); }
  printf(" d_x_(j+1), \n ");
  for(i=0;i<N;i++){
     printf("%e\t",test[i]);
     test1[i]=test[i];
}
  printf("\n");
*/

printf("iteration = %3d, residual = %e\n", k, r1);

alpha=1.0;
beta=0.0;

cusparseStatus1=hipsparseDcsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);
    if(cusparseStatus1!=HIPSPARSE_STATUS_SUCCESS){
    CLEANUP("Ax0 performing failed\n");}

/*
cudaStat1=hipMemcpy(test,  d_Ax, N*sizeof(double), hipMemcpyDeviceToHost);
  if(cudaStat1!=hipSuccess){
    CLEANUP("Memcpy from Device to Host failed\n"); }
  printf(" A*x final, \n ");
  for(i=0;i<N;i++){
    printf("%e\n",test[i]);
  //   test1[i]=test[i];
}
  printf("\n");
*/



cudaStat1=hipMemcpy(test,  d_Ax, N*sizeof(double), hipMemcpyDeviceToHost);
  if(cudaStat1!=hipSuccess){
    CLEANUP("Memcpy from Device to Host failed\n"); }
  printf(" A*x final, \n ");
  for(i=0;i<N;i++){
    fprintf(p1,"%e\n",test[i]);
}
  printf("\n");


cudaStat1=hipMemcpy(test, d_x, N*sizeof(double), hipMemcpyDeviceToHost);
  if(cudaStat1!=hipSuccess){
    CLEANUP("Memcpy from Device to Host failed\n"); }
  printf(" x final, \n ");
  for(i=0;i<N;i++){
     fprintf(p2,"%e\n",test[i]);
     test1[i]=test[i];
}
  printf("\n");




 hipsparseDestroy(cusparseHandle);
    hipblasDestroy(cublasHandle);

    fclose(p1);
    fclose(p2);
    free(I);
    free(J);
    free(val);
    free(r);
    free(test);
    free(hostResults);
    hipFree(d_col);
    hipFree(d_row);
    hipFree(d_val);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ax);
   
    hipDeviceReset();

    


 return test1;

}
}
